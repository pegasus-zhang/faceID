#include "hip/hip_runtime.h"
#include "cuda_utils.h"
// CUDA 核函数：计算 L2 距离
__global__ void computeL2Distance(const cv::cuda::PtrStepSz<float> A, 
                                   const cv::cuda::PtrStepSz<float> B, 
                                   cv::cuda::PtrStepSz<float> D, int m, int n, int d) {
    int rowA = blockIdx.x * blockDim.x + threadIdx.x; // A 的行索引
    int rowB = blockIdx.y * blockDim.y + threadIdx.y; // B 的行索引

    if (rowA < m && rowB < n) {
        float sum = 0.0f;
        for (int i = 0; i < d; ++i) {
            float diff = A(rowA, i) - B(rowB, i); // 计算每一维的差值
            sum += diff * diff;
        }
        D(rowA, rowB) = sqrtf(sum); // 存储 L2 距离到 D 矩阵
    }
}


// **确保 GpuMat 具有连续内存**
void ensureContinuous(cv::cuda::GpuMat& mat) {
    if (!mat.isContinuous()) {
        cv::cuda::GpuMat temp;
        mat.copyTo(temp);  // 复制到新的 GpuMat，确保连续
        mat = temp;
    }
}

// 调用 CUDA 核函数
void computeL2DistanceCUDA(const cv::cuda::GpuMat& A,const cv::cuda::GpuMat& B, cv::cuda::GpuMat& D) {
    int m = A.rows;
    int n = B.rows;
    int d = A.cols;

    D.create(m, n, CV_32F); // 确保结果矩阵已创建

    // 线程块大小 (16, 16)
    dim3 blockSize(16, 16);
    dim3 gridSize((m + blockSize.x - 1) / blockSize.x, (n + blockSize.y - 1) / blockSize.y);

    // 调用 CUDA 核函数
    computeL2Distance<<<gridSize, blockSize>>>(
        A, 
        B, 
        D, 
        m, n, d
    );

    hipDeviceSynchronize();
}

int main() {
    int m = 2, n = 2, d = 2;
    cv::Mat h_A = (cv::Mat_<float>(m, d) << 1, 1, 2, 2);
    cv::Mat h_B = (cv::Mat_<float>(n, d) << 1, 1, 2, 2);
    std::cout << "h_A:\n" << h_A << std::endl;
    std::cout << "h_B:\n" << h_B << std::endl;

    cv::cuda::GpuMat d_A, d_B, d_D;
    d_A.upload(h_A);
    d_B.upload(h_B);

    computeL2DistanceCUDA(d_A, d_B, d_D);

    cv::Mat h_D;
    d_D.download(h_D);

    std::cout << "L2 Distance Matrix:\n" << h_D << std::endl;
    return 0;
}
